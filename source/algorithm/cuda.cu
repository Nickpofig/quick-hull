#include "hip/hip_runtime.h"

// standard
#include <iostream>
#include <sstream>
#include <vector>
#include <cstddef>

// internal
#include "core.hpp"
#include "program.hpp"
#include "algorithm/cuda.hpp"

namespace quick_hull 
{
	// Cuda error handling
	static void cuda_handle_error(hipError_t code, const char * file, int line) 
	{
		if (code == hipSuccess) return; 
		
		program::panic_begin 
			<< "Error(" << code << ") in" << file << " at line " << line 
			<< program::panic_end;
	}


	#define macro_cuda_call(call) (cuda_handle_error(call, __FILE__, __LINE__))


	// Cuda version of vector2 functions
	__device__ Vector2 cuda_vector_make(double x, double y)
	{
		Vector2 vector;
		
		vector.x = x;
		vector.y = y;

		return vector;
	}

	__device__ Vector2 cuda_vector_subtract(const Vector2& a, const Vector2 &b) 
	{
		return cuda_vector_make(a.x - b.x, a.y - b.y);
	}

	__device__ Vector2 cuda_vector_add(const Vector2& a, const Vector2& b) 
	{
		return cuda_vector_make(a.x + b.x, a.y + b.y);
	}

	__device__ Vector2 cuda_vector_get_normal(const Vector2& vector) 
	{
		return cuda_vector_make(-vector.y, vector.x);
	}

	__device__ double cuda_vector_get_sqr_magnitude(const Vector2& vector) 
	{
		return (vector.x * vector.x) + (vector.y * vector.y);
	}
	
	__device__ double cuda_vector_dot_product(const Vector2 &vector_a, const Vector2 &vector_b) 
	{
		return (vector_a.x * vector_b.x) + (vector_a.y * vector_b.y); 
	}

	__device__ Vector2 cuda_vector_project(const Vector2 &vector_a, const Vector2 &vector_b)
	{
		double vector_b_sqr_magnitude = cuda_vector_get_sqr_magnitude(vector_b);

		if (vector_b_sqr_magnitude == 0) 
		{
			return cuda_vector_make(0, 0);
		}
		
		double relativeness = cuda_vector_dot_product(vector_a, vector_b);

		return cuda_vector_make(
			vector_b.x * (relativeness / vector_b_sqr_magnitude), 
			vector_b.y * (relativeness / vector_b_sqr_magnitude)
		);
	}

	struct Cuda_Thread_Data
	{
		public:
			int farest_point_index;
			double farest_point_sqr_distance;
	};

	__global__
	void find_farest_point_from_line
	(
		int* result_point_index, // only the first thread in a block has write access
		const Vector2* points, 
		int point_count, 
		int line_point_a_index, 
		int line_point_b_index
	)
	{
		// Declares dynamic(or not) shared memory
		//extern 
		__shared__ Cuda_Thread_Data block_data[
			1024
		];

		// Defines block settings
		int block_count = gridDim.x;
		int thread_count = blockDim.x;
		int unit_count = thread_count * block_count;

		// Divides point count by number of computation units and ceil it
		int points_per_thread = (point_count + unit_count + 1) / unit_count;
		

		// Defines thread local data
		int block_id = blockIdx.x;
		int thread_id = threadIdx.x;
		int unit_index = thread_count * block_id + thread_id;
		int points_start = (unit_index) * points_per_thread;
		int points_end = (unit_index + 1) * points_per_thread;

		if (point_count < points_end) 
		{
			points_end = point_count;
		}

		double thread_farest_point_sqr_distance = 0;
		int thread_farest_point_index = line_point_a_index;

		Vector2 line_point_a = points[line_point_a_index];
		Vector2 line_point_b = points[line_point_b_index];

		for(int point_index = points_start; point_index < points_end; point_index++) 
		{
			Vector2 point = points[point_index];
			Vector2 line = cuda_vector_subtract(line_point_b, line_point_a);
			Vector2 line_normal = cuda_vector_get_normal(line);

			// projected-point on the line from the current point
			Vector2 projection_base = cuda_vector_add(
				cuda_vector_project(
					cuda_vector_subtract(point, line_point_a), 
					line
				),
				line_point_a 
			); 

			// projection (as vector) from the base to the point
			Vector2 projection = cuda_vector_subtract(point, projection_base);

			double relativity = cuda_vector_dot_product(projection, line_normal);
			double projection_sqr_magnitude = cuda_vector_get_sqr_magnitude(projection);

			if (relativity > 0 && thread_farest_point_sqr_distance < projection_sqr_magnitude)
			{
				thread_farest_point_sqr_distance = projection_sqr_magnitude;
				thread_farest_point_index = point_index;
			}
		}
		
		block_data[thread_id].farest_point_index = thread_farest_point_index;
		block_data[thread_id].farest_point_sqr_distance = thread_farest_point_sqr_distance;

		__syncthreads();

		for (int reduction_step = 2; (thread_id % reduction_step == 0) && (thread_count >= reduction_step); reduction_step <<= 1)
		{
			int supply_thread_id = thread_id + (reduction_step >> 1);

			if (supply_thread_id >= thread_count) break;

			if (block_data[thread_id].farest_point_sqr_distance < block_data[supply_thread_id].farest_point_sqr_distance) 
			{
				block_data[thread_id].farest_point_sqr_distance = block_data[supply_thread_id].farest_point_sqr_distance;
				block_data[thread_id].farest_point_index = block_data[supply_thread_id].farest_point_index;
			};

			__syncthreads();
		}

		if (thread_id == 0) 
		{
			result_point_index[block_id] = block_data[0].farest_point_index;
		}
	}

	// Run over all points
	// Finds farest point from a given line
	// Add the point to convex hull

	template<int T_Block_Count, int T_Thread_Count>
	void grow
	(
		int point_a_index, 
		int point_b_index, 
		const std::vector<Vector2> & points,
		std::vector<Vector2> * convex_hull,
		Vector2* device_points_copy,
		int* host_far_points,
		int* device_far_points
	)
	{
		find_farest_point_from_line<<<T_Block_Count, T_Thread_Count>>>
		(
			device_far_points,
			device_points_copy,
			points.size(),
			point_a_index, 
			point_b_index
		);

		macro_cuda_call(hipDeviceSynchronize());

		size_t far_points_memsize = 1 * sizeof(int);
		macro_cuda_call(hipMemcpy(host_far_points, device_far_points, far_points_memsize, hipMemcpyDeviceToHost));

		int point_c_index = host_far_points[0];

		if (point_c_index != point_a_index && point_c_index != point_b_index)
		{
			// a convex hull from the AC line 
			grow<T_Block_Count, T_Thread_Count>(
				point_a_index, 
				point_c_index, 
				points, 
				convex_hull, 
				device_points_copy,
				host_far_points,
				device_far_points
			); 

			convex_hull->push_back(points[point_c_index]);

			// a convex hull from the CB line
			grow(
				point_c_index, 
				point_b_index, 
				points, 
				convex_hull
				device_points_copy,
				host_far_points,
				device_far_points
			); 
		}
	}




	Quick_Hull_Cuda::~Quick_Hull_Cuda() { }

	std::vector<Vector2> * Quick_Hull_Cuda::run(const std::vector<Vector2> &points)
	{
		int most_left_index  = 0;
		int most_right_index = points.size() - 1;
		
		// Finds the most left and right point
		for (int index = 0; index < points.size(); index++)
		{
			const auto point = points[index];
			const auto most_right = points[most_right_index];
			const auto most_left = points[most_left_index];

			if (point.x > most_right.x || (point.x == most_right.x && point.y > most_right.y))
			{
				most_right_index = index;
			}
			else
			if (point.x < most_left.x || (point.x == most_right.x && point.y < most_right.y)) 
			{
				most_left_index = index;
			}
		}

		auto *convex_hull = new std::vector<Vector2>();

		// Initializes cuda working memory
		size_t far_points_memsize = 1 * sizeof(int);
		size_t points_memsize = points.size() * sizeof(Vector2);

		macro_cuda_call(hipHostAlloc((void**) &host_far_points, far_points_memsize, hipHostMallocDefault));
		macro_cuda_call(hipMalloc((void**) &device_far_points, far_points_memsize));
		macro_cuda_call(hipMalloc((void**) &device_points_copy, points_memsize));
		macro_cuda_call(hipMemcpy(device_points_copy, points.data(), points_memsize, hipMemcpyHostToDevice));


		// Constructs a convex from right and left side of line going through the most left and right points

		convex_hull->push_back(points[most_left_index]);
		
		grow<>(
			most_left_index, 
			most_right_index, 
			points, 
			convex_hull, 
			device_points_copy,
			host_far_points,
			device_far_points
		);
		
		convex_hull->push_back(points[most_right_index]);
		
		grow(
			most_right_index, 
			most_left_index,
			points, 
			convex_hull, 
			device_points_copy,
			host_far_points,
			device_far_points
		);


		macro_cuda_call(hipHostFree(host_far_points));
		macro_cuda_call(hipFree(device_far_points));
		macro_cuda_call(hipFree(device_points_copy));

		return convex_hull;
	}
}

