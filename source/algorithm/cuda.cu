#include "hip/hip_runtime.h"

// standard
#include <iostream>
#include <sstream>
#include <vector>
#include <cstddef>

// internal
#include "core.hpp"
#include "program.hpp"
#include "algorithm/cuda.hpp"

namespace quick_hull 
{
	// Cuda error handling
	static void cuda_handle_error(hipError_t error, const char * file, int line) 
	{
		if (error == hipSuccess) return; 
		
		program::panic_begin 
			<< "Error(" << error << ") in" << file << " at line " << line 
			<< program::panic_end;
	}


	#define macro_cuda_call(call) (cuda_handle_error(call, __FILE__, __LINE__))


	// Cuda version of vector2 functions
	__device__ Vector2 cuda_vector_make(const double x, const double y)
	{
		Vector2 vector;
		
		vector.x = x;
		vector.y = y;

		return vector;
	}

	__device__ Vector2 cuda_vector_subtract(const Vector2& a, const Vector2 &b) 
	{
		return cuda_vector_make(a.x - b.x, a.y - b.y);
	}

	__device__ Vector2 cuda_vector_add(const Vector2& a, const Vector2& b) 
	{
		return cuda_vector_make(a.x + b.x, a.y + b.y);
	}

	__device__ Vector2 cuda_vector_get_normal(const Vector2& vector) 
	{
		return cuda_vector_make(-vector.y, vector.x);
	}

	__device__ double cuda_vector_get_sqr_magnitude(const Vector2& vector) 
	{
		return (vector.x * vector.x) + (vector.y + vector.y);
	}
	
	__device__ double cuda_vector_dot_product(const Vector2 &vector_a, const Vector2 &vector_b) 
	{
		return (vector_a.x * vector_b.x) + (vector_a.y * vector_b.y); 
	}

	__device__ Vector2 cuda_vector_project(const Vector2 &vector_a, const Vector2 &vector_b)
	{
		double vector_b_sqr_magnitude = cuda_vector_get_sqr_magnitude(vector_b);

		if (vector_b_sqr_magnitude == 0) 
		{
			return cuda_vector_make(0, 0);
		}
		
		double relativeness = cuda_vector_dot_product(vector_a, vector_b);

		return cuda_vector_make(
			vector_b.x * (relativeness / vector_b_sqr_magnitude), 
			vector_b.y * (relativeness / vector_b_sqr_magnitude)
		);
	}

	struct Cuda_Thread_Data
	{
		public:
			int farest_point_index;
			int farest_point_sqr_distance;
	};

	__global__
	void find_farest_point_from_line
	(
		int* result_point_index, // only the first thread in a block has write access
		const Vector2* points, 
		int point_count, 
		Vector2 line_point_a, 
		Vector2 line_point_b
	)
	{
		// Declares dynamic(or not) shared memory
		//extern 
		__shared__ Cuda_Thread_Data block_data[
			1024
		];

		// Defines block settings
		int block_count = gridDim.x;
		int thread_count = blockDim.x;
		int unit_count = thread_count * block_count;

		// Divides point count by number of computation units and ceil it
		int points_per_thread = (point_count + unit_count + 1) / unit_count;
		

		// Defines thread local data
		int block_id = blockIdx.x;
		int thread_id = threadIdx.x;
		int unit_index = thread_count * block_id + thread_id;
		int points_start = (unit_index) * points_per_thread;
		int points_end = (unit_index + 1) * points_per_thread;

		if (point_count < points_end) 
		{
			points_end = point_count;
		}

		double thread_farest_point_sqr_distance = ~0; // must be the minimum number in two's complement system
		int thread_farest_point_index = points_start;

		for(int point_index = points_start; point_index < points_end; point_index++) 
		{
			Vector2 point = points[point_index];
			Vector2 line = cuda_vector_subtract(line_point_a, line_point_b);
			Vector2 line_normal = cuda_vector_get_normal(line);

			// projected-point on the line from the current point
			Vector2 projection_base = cuda_vector_project(
				cuda_vector_subtract(point, line_point_a), 
				cuda_vector_add(line, line_point_a)
			); 

			// projection (as vector) from the base to the point
			Vector2 projection = cuda_vector_subtract(point, projection_base);

			double relativity = cuda_vector_dot_product(projection, line_normal);
			double projection_sqr_magnitude = cuda_vector_get_sqr_magnitude(projection);

			if (relativity < 0)
			{
				projection_sqr_magnitude = -projection_sqr_magnitude;
			}
			
			if (thread_farest_point_sqr_distance < projection_sqr_magnitude) 
			{
				thread_farest_point_sqr_distance = projection_sqr_magnitude;
				thread_farest_point_index = point_index;
			}
		}
		
		block_data[thread_id].farest_point_index = thread_farest_point_index;
		block_data[thread_id].farest_point_sqr_distance = thread_farest_point_sqr_distance;

		__syncthreads();

		for (int reduction_step = 2; (thread_id % reduction_step == 0) && (thread_count >= reduction_step); reduction_step <<= 1)
		{
			int supply_thread_id = thread_id + (reduction_step >> 1);

			if (supply_thread_id >= thread_count) break;

			if (block_data[thread_id].farest_point_sqr_distance < block_data[supply_thread_id].farest_point_sqr_distance) 
			{
				block_data[thread_id].farest_point_sqr_distance = block_data[supply_thread_id].farest_point_sqr_distance;
				block_data[thread_id].farest_point_index = block_data[supply_thread_id].farest_point_index;
			};

			__syncthreads();
		}

		if (thread_id == 0) 
		{
			result_point_index[block_id] = block_data[0].farest_point_index;
		}
	}

	// Run over all points
	// Finds farest point from a given line
	// Add the point to convex hull

	void grow
	(
		Vector2 a, 
		Vector2 b, 
		const std::vector<Vector2> & points,
		std::vector<Vector2> * convex_hull
	)
	{
		int* host_far_points;
		int* device_far_points;
		Vector2* device_points_copy;

		size_t far_points_memsize = 1 * sizeof(int);
		size_t points_memsize = points.size() * sizeof(Vector2);


		program::log_begin << "Allocating device and host memory... points memsize: " << points_memsize << "." << program::log_end;

		macro_cuda_call(hipHostAlloc((void**) &host_far_points, far_points_memsize, hipHostMallocDefault));
		macro_cuda_call(hipMalloc((void**) &device_far_points, far_points_memsize));
		macro_cuda_call(hipMalloc((void**) &device_points_copy, points_memsize));

		macro_cuda_call(hipMemcpy(device_points_copy, points.data(), points_memsize, hipMemcpyHostToDevice));


		program::log_begin << "Executing kernel..." << program::log_end;

		find_farest_point_from_line<<<1, 1024
		//, 1024
		>>>
		(
			device_far_points,
			device_points_copy,
			points.size(),
			a,
			b
		);

		macro_cuda_call(hipDeviceSynchronize());
		macro_cuda_call(hipMemcpy(host_far_points, device_far_points, far_points_memsize, hipMemcpyDeviceToHost));

		program::log_begin << "Reading result point..." << program::log_end;

		Vector2 c = points.at(host_far_points[0]);

		macro_cuda_call(hipHostFree(host_far_points));
		macro_cuda_call(hipFree(device_far_points));
		macro_cuda_call(hipFree(device_points_copy));

		if (c != a && c != b)
		{
			program::log_begin << "Growing convex hull left..." << program::log_end;
			grow(a, c, points, convex_hull); // a convex hull from the AC line 

			program::log_begin << "Adding convex hull point..." << program::log_end;
			convex_hull->push_back(c);

			program::log_begin << "Growing convex hull right..." << program::log_end;
			grow(c, b, points, convex_hull); // a convex hull from the CB line
		}
	}




	Quick_Hull_Cuda::~Quick_Hull_Cuda() { }

	std::vector<Vector2> * Quick_Hull_Cuda::run(const std::vector<Vector2> &points)
	{
		Vector2 most_left  = points.front();
		Vector2 most_right = points.back();
		
		// Finds the most left and right point
		for (const auto &point: points)
		{
			if (point.x > most_right.x || (point.x == most_right.x && point.y > most_right.y))
			{
				most_right = point;
			}
			else
			if (point.x < most_left.x || (point.x == most_right.x && point.y < most_right.y)) 
			{
				most_left  = point;
			}
		}

		// Convex hull 
		auto *convex_hull = new std::vector<Vector2>();

		// Constructs a convex from right and left side of line going through the most left and right points
		grow(most_left, most_right, points, convex_hull);
		grow(most_right, most_left, points, convex_hull);

		return convex_hull;
	}
}

