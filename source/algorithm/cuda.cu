#include "hip/hip_runtime.h"

// standard
#include <iostream>
#include <sstream>
#include <vector>
#include <cstddef>
#include <chrono>

// internal
#include "core.hpp"
#include "console.hpp"
#include "algorithm/cuda.hpp"

namespace quick_hull 
{
	// CUDA error handling
	static void cuda_handle_error(hipError_t code, const char * file, int line) 
	{
		if (code == hipSuccess) return; 
		
		program::panic_begin 
			<< "Error(" << code << ") in" << file << " at line " << line 
			<< program::panic_end;
	}

	#define macro_cuda_call(call) (cuda_handle_error(call, __FILE__, __LINE__))


	// CUDA version of vector2 functions
	__device__ Vector2 cuda_vector_make(double x, double y)
	{
		Vector2 vector;
		
		vector.x = x;
		vector.y = y;

		return vector;
	}

	__device__ Vector2 cuda_vector_subtract(const Vector2& a, const Vector2 &b) 
	{
		return cuda_vector_make(a.x - b.x, a.y - b.y);
	}

	__device__ Vector2 cuda_vector_add(const Vector2& a, const Vector2& b) 
	{
		return cuda_vector_make(a.x + b.x, a.y + b.y);
	}

	__device__ Vector2 cuda_vector_get_normal(const Vector2& vector) 
	{
		return cuda_vector_make(-vector.y, vector.x);
	}

	__device__ double cuda_vector_get_sqr_magnitude(const Vector2& vector) 
	{
		return (vector.x * vector.x) + (vector.y * vector.y);
	}
	
	__device__ double cuda_vector_dot_product(const Vector2 &vector_a, const Vector2 &vector_b) 
	{
		return (vector_a.x * vector_b.x) + (vector_a.y * vector_b.y); 
	}

	__device__ Vector2 cuda_vector_project(const Vector2 &vector_a, const Vector2 &vector_b)
	{
		double vector_b_sqr_magnitude = cuda_vector_get_sqr_magnitude(vector_b);

		if (vector_b_sqr_magnitude == 0) 
		{
			return cuda_vector_make(0, 0);
		}
		
		double relativeness = cuda_vector_dot_product(vector_a, vector_b);

		return cuda_vector_make(
			vector_b.x * (relativeness / vector_b_sqr_magnitude), 
			vector_b.y * (relativeness / vector_b_sqr_magnitude)
		);
	}




	__global__
	void find_farest_point_from_line
	(
		Cuda_Thread_Data* result_point_index, // only the first thread in a block has write access
		const Vector2* points, 
		int point_count, 
		int line_point_a_index, 
		int line_point_b_index
	)
	{
		// Declares the block memory
		__shared__ Cuda_Thread_Data block_data
		[
			1024 // ..the maximum amount of threads
		];

		// Defines dimensions
		int block_count = gridDim.x;
		int thread_count = blockDim.x;
		int unit_count = thread_count * block_count;

		// Divides point count by number of computation units and ceil it to get a per thread standard amount of points
		int points_per_thread = (point_count + unit_count + 1) / unit_count;

		// Defines thread local data
		int block_id = blockIdx.x;
		int thread_id = threadIdx.x;
		int unit_index = thread_count * block_id + thread_id;
		int points_start = (unit_index) * points_per_thread;
		int points_end = (unit_index + 1) * points_per_thread;

		// Amount of points may not be a power of two. 
		// That is why, it is necessary to trim points_end of the last thread
		if (points_end > point_count) // ..no need to check a thread id, because only the last one could satisfy this condition
		{
			points_end = point_count;
		}

		// By default, for each thread, sets point A as the farest one
		block_data[thread_id].farest_point_index = line_point_a_index;
		block_data[thread_id].farest_point_sqr_distance = 0;

		// Defines AB line and its normal towards the considered side
		Vector2 line_point_a = points[line_point_a_index];
		Vector2 line_point_b = points[line_point_b_index];
		Vector2 line = cuda_vector_subtract(line_point_b, line_point_a);
		Vector2 line_normal = cuda_vector_get_normal(line);

		// 1. Finds the farest point from AB line
		for(int point_index = points_start; point_index < points_end; point_index++) 
		{
			Vector2 point = points[point_index];

			// A point, which is a base of a projection on the AB line from the current point
			Vector2 projection_base = cuda_vector_add(
				cuda_vector_project(
					cuda_vector_subtract(point, line_point_a), 
					line
				),
				line_point_a 
			); 

			// The projection (as vector) from the base to the point
			Vector2 projection = cuda_vector_subtract(point, projection_base);

			// Relativity checks that point is on the considered side of the AB line
			double relativity = cuda_vector_dot_product(projection, line_normal);
			double projection_sqr_magnitude = cuda_vector_get_sqr_magnitude(projection);

			// Saves the farest point data
			if (relativity > 0 && block_data[thread_id].farest_point_sqr_distance < projection_sqr_magnitude)
			{
				block_data[thread_id].farest_point_index = point_index;
				block_data[thread_id].farest_point_sqr_distance = projection_sqr_magnitude;
			}
		}
		
		__syncthreads();


		// 2. Reduces a result of all threads using the (binary / a power of two) reduction method. 
		//    The farest point will be stored in the first thread memory.
		for (int reduction_step = 2; (thread_id % reduction_step == 0) && (thread_count >= reduction_step); reduction_step <<= 1)
		{
			// Gets a position of a new subject thread.
			int subject_thread_id = thread_id + (reduction_step >> 1);

			if (subject_thread_id >= thread_count) break; // .. there is no subject thread left

			// Checks for a subject thread to have a farther point and overrides this thread data when it does.
			if (block_data[thread_id].farest_point_sqr_distance < block_data[subject_thread_id].farest_point_sqr_distance) 
			{
				block_data[thread_id].farest_point_sqr_distance = block_data[subject_thread_id].farest_point_sqr_distance;
				block_data[thread_id].farest_point_index = block_data[subject_thread_id].farest_point_index;
			};

			__syncthreads();
		}

		// 3. Copies the result of the first thread into thread's block result.
		if (thread_id == 0)
		{
			auto* copy_to   = &result_point_index[block_id];
			auto* copy_from = &block_data[0];

			copy_to->farest_point_index        = copy_from->farest_point_index;
			copy_to->farest_point_sqr_distance = copy_from->farest_point_sqr_distance;
		}
	}




	// Implements [ grow ] function
	template<int T_Block_Count, int T_Thread_Count>
	void Algorithm_Cuda::grow
	(
		int point_a_index, 
		int point_b_index, 
		const std::vector<Vector2> & points
	)
	{
		// Captures kernel start time
		auto stopwatch_start = std::chrono::steady_clock::now();

		// 1. Runs the kernel function, which finds far points along the AB line
		find_farest_point_from_line<<<T_Block_Count, T_Thread_Count>>>
		(
			device_far_points,
			device_points_copy,
			points.size(),
			point_a_index, 
			point_b_index
		);

		// Waits the kernel function
		macro_cuda_call(hipDeviceSynchronize());

		// Captures kernel end time
		auto stopwatch_end = std::chrono::steady_clock::now();

		// Gathers analytic data
		this->total_recursion_call_count++; // ..recurison counting
		this->kernel_total_time += std::chrono::duration<double, std::milli> // ..kernel ellapsed milliseconds
		(
			stopwatch_end - stopwatch_start
		)
		.count(); 


		// 2. Copies the result data from the device to the host
		size_t far_points_memsize = T_Block_Count * sizeof(Cuda_Thread_Data);
		stopwatch_start = std::chrono::steady_clock::now();

		macro_cuda_call(hipMemcpy(host_far_points, device_far_points, far_points_memsize, hipMemcpyDeviceToHost));

		stopwatch_end = std::chrono::steady_clock::now();
		this->cuda_memcpy_total_time += std::chrono::duration<double, std::milli>
		(
			stopwatch_end - stopwatch_start
		)
		.count(); 


		// 3. Reduces result of blocks into the first block memory.
		//    The farest point among others will be stored.

		stopwatch_start = std::chrono::steady_clock::now();
		for (int step = 1; step < T_Block_Count; step <<= 1) 
		{
			for (int index = 0; index < T_Block_Count; index += (step << 1)) 
			{
				auto* recepient = &host_far_points[index];
				auto* donor = &host_far_points[index + step];

				if (recepient->farest_point_sqr_distance < donor->farest_point_sqr_distance)
				{
					recepient->farest_point_index = donor->farest_point_index;
					recepient->farest_point_sqr_distance = donor->farest_point_sqr_distance;
				}
			}
		}

		stopwatch_end = std::chrono::steady_clock::now();
		this->reduction_total_time += std::chrono::duration<double, std::milli>
		(
			stopwatch_end - stopwatch_start
		)
		.count();

		// Gets the farest point from AB line
		int point_c_index = host_far_points[0].farest_point_index;

		// Checks it to be different from A and B point
		if (point_c_index != point_a_index && point_c_index != point_b_index)
		{
			// Tries to grow the convex hull from the AC line 
			grow<T_Block_Count, T_Thread_Count>(
				point_a_index, 
				point_c_index, 
				points
			); 

			// Adds the founded farest point to the convex hull
			convex_hull->push_back(points[point_c_index]);

			// Tries to grow the convex hull from the CB line
			grow<T_Block_Count, T_Thread_Count>(
				point_c_index, 
				point_b_index, 
				points
			); 
		}
	}

	template<int T_Block_Count, int T_Thread_Count>
	std::vector<Vector2> * Algorithm_Cuda::internal_run
	(
		const std::vector<Vector2> &points
	)
	{
		int most_left_index  = 0;
		int most_right_index = points.size() - 1;
		
		// 1. Finds the most left and right point
		for (int index = 0; index < points.size(); index++)
		{
			const auto point = points[index];
			const auto most_right = points[most_right_index];
			const auto most_left = points[most_left_index];

			if (point.x > most_right.x || (point.x == most_right.x && point.y > most_right.y))
			{
				most_right_index = index;
			}
			else
			if (point.x < most_left.x || (point.x == most_right.x && point.y < most_right.y)) 
			{
				most_left_index = index;
			}
		}
		
		size_t far_points_memsize = (T_Block_Count) * sizeof(Cuda_Thread_Data);
		size_t points_memsize = points.size() * sizeof(Vector2);

		// 2. Allocates special CUDA memory

		auto stopwatch_start = std::chrono::steady_clock::now();
		
		// ============================================================
		// !!! This CALL is 60~80 % of the program execution time !!!!!
		// !!! Makes CUDA implementation massively inefficient !!!!!!!!
		macro_cuda_call(hipHostAlloc((void**) &host_far_points, far_points_memsize, hipHostMallocDefault));
		macro_cuda_call(hipMalloc((void**) &device_far_points, far_points_memsize));
		macro_cuda_call(hipMalloc((void**) &device_points_copy, points_memsize));
		macro_cuda_call(hipMemcpy(device_points_copy, points.data(), points_memsize, hipMemcpyHostToDevice));
		// ============================================================

		auto stopwatch_end = std::chrono::steady_clock::now();
		this->cuda_meminit_total_time = std::chrono::duration<double, std::milli>
		(
			stopwatch_end - stopwatch_start
		)
		.count();


		// 3. Constructs a convex from right and left side of line going through the most left and right points.

		convex_hull = new std::vector<Vector2>();

		convex_hull->push_back(points[most_left_index]);
		
		// 3.1 Grows the convex hull from ML->MR line 
		grow<T_Block_Count, T_Thread_Count>(
			most_left_index, 
			most_right_index, 
			points
		);
		
		convex_hull->push_back(points[most_right_index]);
		
		// 3.2 Grows the convex hull from MR->ML line 
		grow<T_Block_Count, T_Thread_Count>(
			most_right_index, 
			most_left_index,
			points
		);

		// 4. Releases special CUDA memory
		macro_cuda_call(hipHostFree(host_far_points));
		macro_cuda_call(hipFree(device_far_points));
		macro_cuda_call(hipFree(device_points_copy));

		return convex_hull;
	}




	Algorithm_Cuda::~Algorithm_Cuda() { }

	Algorithm_Cuda::Algorithm_Cuda(int block_power) 
	{
		this->block_power = block_power;
	}

	std::vector<Vector2> * Algorithm_Cuda::run(const std::vector<Vector2> &points)
	{
		// Resets analytic data
		this->kernel_total_time = 0;
		this->total_recursion_call_count = 0;
		this->cuda_memcpy_total_time = 0;
		this->reduction_total_time = 0;

		// Depending on the power of the number of CUDA blocks,
		// matches with a call, with a correct number of threads per block
		switch(this->block_power) 
		{
			// x1
			case 0 : return internal_run<1,1024>(points);
			// x2
			case 1 : return internal_run<2,512>(points);
			// x4
			case 2 : return internal_run<4,256>(points);
			// x8
			case 3 : return internal_run<8,128>(points);
			// x16
			default: // by default, if block power has not been set then 16 blocks and 64 threads will be used
			case 4 : return internal_run<16,64>(points);
			// x32
			case 5 : return internal_run<32,32>(points); 
			// x64
			case 6 : return internal_run<64,16>(points); 
			// x128
			case 7 : return internal_run<128,8>(points);
			// x256
			case 8 : return internal_run<256,4>(points);
			// x512
			case 9 : return internal_run<512,2>(points);
			// x1024
			case 10: return internal_run<1024,1>(points);
		}
	}
}

